#include "row.cuh"

void compute::gpu::row::reset(unsigned long length)
{
	init = false; cleanup();

	this->length = length;

	if (hipHostMalloc((void**)&data, length * sizeof(int)) != hipSuccess) return;
	if (data == NULL) return;

	clear();

	init = true;
}

void compute::gpu::row::clear()
{
	for (unsigned long i = 0UL; i < length; ++i) data[i] = 0;

	top.clear();
}

int compute::gpu::row::get(unsigned long idx)
{
	if (idx >= length) return -1;

	return data[idx];
}

void compute::gpu::row::set(header *source)
{
	top = *source;
}

bool compute::gpu::row::set(unsigned long idx)
{
	if (idx >= length) return false;

	data[idx] = 1;

	return true;
}

bool compute::gpu::row::copy(row const &source)
{
	bool result = true;
	if (length != source.length) result = false;

	clear();

	unsigned long temp = source.length;
	if (temp >= length) temp = length;

	for (unsigned long i = 0UL; i < temp; ++i)
	{
		data[i] = source.data[i];
	}

	top = source.top;

	return result;
}

void compute::gpu::row::makeNull()
{
	data = NULL;
}

void compute::gpu::row::cleanup()
{
	if (data != NULL) hipHostFree(data);
}