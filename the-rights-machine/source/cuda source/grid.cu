#include "hip/hip_runtime.h"
#include "grid.cuh"
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include "custom/string.h"
#include "log.h"

unsigned long compute::gpu::grid::GRIDS = 255;
unsigned long compute::gpu::grid::THREADS = 255;

unsigned long compute::gpu::grid::WIDTH = 255;
unsigned long compute::gpu::grid::HEIGHT = 255;

__global__ void minusKernel(int *a, const int *b)
{
	int i = threadIdx.x;

	a[i] = a[i] - b[i];
}

__global__ void andKernel(int *a, const int *b)
{
	int i = threadIdx.x;

	a[i] = a[i] & b[i];
}

__global__ void compareKernel(const int *a, const int *b, int *result)
{
	int i = threadIdx.x;

	result[i] = ((!(b[i] & 0x1)) & (a[i] & 0x1));
}

void compute::gpu::grid::reset(unsigned long width, unsigned long height)
{
	init = false; cleanup();

	this->width = width;
	this->height = height;

	write_ptr = 0UL;

	headers = new header*[height];
	if (headers == NULL) return;
	for (unsigned long i = 0UL; i < height; ++i) headers[i] = NULL;

	for (unsigned long i = 0UL; i < height; ++i)
	{
		headers[i] = new header();
		if (headers[i] == NULL) return;
	}

	if (!hipMalloc((void**)&data, width * height * sizeof(int))) return;
	if (!hipMalloc((void**)&temp, width * height * sizeof(int))) return;

	clear();

	init = true;
}

void compute::gpu::grid::clear()
{
	write_ptr = 0UL;
	hipMemset(data, 0, width * height * sizeof(int));
	for (unsigned long i = 0UL; i < height; ++i) headers[i]->clear();
}

bool compute::gpu::grid::isempty()
{
	for (unsigned long i = 0UL; i < height; ++i)
	{
		if (!headers[i]->isempty()) return false;
	}

	return true;
}

void compute::gpu::grid::minus(grid &right)
{
	minusKernel<<<GRIDS, THREADS>>>(data, right.data);
}

void compute::gpu::grid::AND(grid &right)
{
	andKernel<<<GRIDS, THREADS>>>(data, right.data);
}

bool compute::gpu::grid::compare(grid &right)
{
	compareKernel<<<GRIDS, THREADS>>>(data, right.data, temp);

	thrust::device_ptr<int> t_ptr = thrust::device_pointer_cast(temp);
	int result = thrust::reduce(thrust::device, t_ptr, t_ptr + (width * height));

	return result == 0;	
}

bool compute::gpu::grid::push(compute::common::row *source)
{
	if (source->count() > width) return false;
	if (write_ptr >= height) return false;

	unsigned long offset = (write_ptr * width);

	*headers[write_ptr] = source->first();

	if (hipMemcpy(&data[offset], source->raw(), source->count() * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice) != hipSuccess) return false;

	++write_ptr;

	return true;
}

void compute::gpu::grid::output()
{
	int *temp = new int[width * height];
	if (temp == NULL) return;

	if (hipMemcpy(temp, data, width * height * sizeof(int), hipMemcpyKind::hipMemcpyDeviceToHost) != hipSuccess)
	{
		for (unsigned long i = 0UL; i < height; ++i)
		{
			if (!headers[i]->isempty())
			{
				string result = headers[i]->serialize();

				result += ",\"row\":{";
				if (temp[0] > 0) result += "\"0\":1";

				for (unsigned long j = 1UL; j < width; ++j)
				{
					if (temp[j] > 0) result += ",\"" + string::fromInt((int)j) + "\":1";
				}

				result += "}";

				Log << result << string("\r\n");
			}
		}
	}

	delete temp;
}

void compute::gpu::grid::makeNull()
{
	headers = NULL;
	data = NULL;
	temp = NULL;
}

void compute::gpu::grid::cleanup()
{
	if (headers != NULL)
	{
		for (long i = (height - 1L); i >= 0L; i--)
		{
			if (headers[i] != NULL) delete headers[i];
		}

		delete headers;
	}

	if (temp != NULL) hipFree(temp);
	if (data != NULL) hipFree(data);
}
