#include "hip/hip_runtime.h"
#include "grid.cuh"
#include "log.h"

__global__ void minusKernel(int *a, const int *b)
{
	int i = threadIdx.x;

	a[i] = a[i] - b[i];
}

__global__ void andKernel(int *a, const int *b)
{
	int i = threadIdx.x;

	a[i] = a[i] & b[i];
}

void compute::gpu::grid::reset(unsigned long width, unsigned long height)
{
	init = false; cleanup();

	this->width = width;
	this->height = height;

	write_ptr = 0UL;

	headers = new header*[height];
	if (headers == NULL) return;
	for (unsigned long i = 0UL; i < height; ++i) headers[i] = NULL;

	for (unsigned long i = 0UL; i < height; ++i)
	{
		headers[i] = new header();
		if (headers[i] == NULL) return;
	}

	if (!hipMalloc((void**)&data, width * height * sizeof(int))) return;

	clear();

	init = true;
}

void compute::gpu::grid::clear()
{
	write_ptr = 0UL;
	memset(data, 0, sizeof(int) * width * height);
	for (unsigned long i = 0UL; i < height; ++i) headers[i]->clear();
}

bool compute::gpu::grid::isempty()
{
	for (unsigned long i = 0UL; i < height; ++i)
	{
		if (!headers[i]->isempty()) return false;
	}

	return true;
}

void compute::gpu::grid::minus(grid &right)
{
	minusKernel<<<GRIDS, THREADS>>>(data, right.data);
}

void compute::gpu::grid::and(grid &right)
{
	andKernel<<<GRIDS, THREADS>>>(data, right.data);
}

bool compute::gpu::grid::compare(grid &right)
{
	// reduce kernel here, need temp???
	//unsigned long offset = 0UL;

	/*
	for (unsigned long y = 0UL; y < height; ++y)
	{
		for (unsigned long x = 0UL; x < width; ++x)
		{
			if ((right.data[offset + x] == 1) && (data[offset + x] == 0)) return false;
		}

		offset += width;
	}
	*/
	return true;
}

bool compute::gpu::grid::push(row &source)
{
	if (source.length > width) return false;
	if (write_ptr >= height) return false;

	unsigned long offset = (write_ptr * width);

	*headers[write_ptr] = source.top;

	if (hipMemcpy(&data[offset], source.data, source.length * sizeof(int), hipMemcpyKind::hipMemcpyHostToDevice) != hipSuccess) return false;

	++write_ptr;

	return true;
}

void compute::gpu::grid::output()
{
	for (unsigned long i = 0UL; i < height; ++i)
	{
		if (!headers[i]->isempty())
		{
			string result = headers[i]->serialize();

			result += ",\"row\":{";
			if (data[0] > 0) result += "\"0\":1";

			for (unsigned long j = 1UL; j < width; ++j)
			{
				if (data[j] > 0) result += ",\"" + string::fromInt((int)j) + "\":1";
			}

			result += "}";

			Log << result << "\r\n";
		}
	}
}

void compute::gpu::grid::makeNull()
{
	headers = NULL;
	data = NULL;
}

void compute::gpu::grid::cleanup()
{
	if (headers != NULL)
	{
		for (long i = (height - 1L); i >= 0L; i--)
		{
			if (headers[i] != NULL) delete headers[i];
		}

		delete headers;
	}

	if (data != NULL) hipFree(data);
}
