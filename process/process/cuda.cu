#include "cuda.cuh"

long compute::cuda::cuda::INSTANCES = 0L;

compute::cuda::cuda::cuda()
{
	if(INSTANCES == 0L) hipSetDevice(0);
}

compute::cuda::cuda::~cuda()
{
	if (--INSTANCES == 0L) hipDeviceReset();
}